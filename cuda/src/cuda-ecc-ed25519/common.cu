
#ifndef COMMON_CU
#define COMMON_CU

static __host__ __device__ uint64_t load_3(const unsigned char *in) {
    uint64_t result;

    result = (uint64_t) in[0];
    result |= ((uint64_t) in[1]) << 8;
    result |= ((uint64_t) in[2]) << 16;

    return result;
}

static __host__ __device__ uint64_t load_4(const unsigned char *in) {
    uint64_t result;

    result = (uint64_t) in[0];
    result |= ((uint64_t) in[1]) << 8;
    result |= ((uint64_t) in[2]) << 16;
    result |= ((uint64_t) in[3]) << 24;
    
    return result;
}

#endif
